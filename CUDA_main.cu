// [[Rcpp::depends(RcppArmadillo)]]
// [[Rcpp::depends(RcppEigen)]]


#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>
using namespace std;

//======================================
//         Grids
//======================================

void gridx(const int nx, const double xmin, const double xmax, double* xgrid){

  const double size = nx;
  const double xstep = (xmax - xmin) /(size - 1);
  double it = 0;

  for(int i = 0; i < nx; i++){
    xgrid[i] = xmin + it*xstep;
    it++;
  }
}


void gride(const int ne, const double ssigma_eps, const double llambda_eps, const double m, double* egrid){

  // This grid is made with Tauchen (1986)
  const double size = ne;
  const double ssigma_y = sqrt(pow(ssigma_eps, 2) / (1 - pow(llambda_eps, 2)));
  const double estep = 2*ssigma_y*m / (size-1);
  double it = 0;

  for(int i = 0; i < ne; i++){
    egrid[i] = (-m*sqrt(pow(ssigma_eps, 2) / (1 - pow(llambda_eps, 2))) + it*estep);
    it++;
  }
}

double normCDF(const double value){
  return 0.5 * erfc(-value * M_SQRT1_2);
}



void eprob(const int ne, const double ssigma_eps, const double llambda_eps, const double m, const double* egrid, double* P){

  // This grid is made with Tauchen (1986)
  // P is: first ne elements are transition from e_0 to e_i,
  //       second ne elementrs are from e_1 to e_i, ...
  const double w = egrid[1] - egrid[0];

  for(int j = 0; j < ne; j++){
    for(int k = 0; k < ne; k++){
      if(k == 0){
        P[j*ne + k] = normCDF((egrid[k] - llambda_eps*egrid[j] + (w/2))/ssigma_eps);
      } else if(k == ne-1){
        P[j*ne + k] = 1 - normCDF((egrid[k] - llambda_eps*egrid[j] - (w/2))/ssigma_eps);
      } else{
        P[j*ne + k] = normCDF((egrid[k] - llambda_eps*egrid[j] + (w/2))/ssigma_eps) - normCDF((egrid[k] - llambda_eps*egrid[j] - (w/2))/ssigma_eps);
      }
    }
  }
}



//======================================
//         Parameter structure
//======================================

class parameters{
 public:
  int nx;
  double xmin;
  double xmax;
  int ne;
  double ssigma_eps;
  double llambda_eps;
  double m;

  double ssigma;
  double eeta;
  double ppsi;
  double rrho;
  double llambda;
  double bbeta;
  int T;
  double r;
  double w;

  void load(const char*);
};



//======================================
//         MAIN  MAIN  MAIN
//======================================

__global__ void Vmaximization(const parameters params, const double* xgrid, const double* egrid, const double* P, const int age, double* V){

  // Recover the parameters
  const int nx              = params.nx;
  const int ne              = params.ne;
  const double ssigma        = params.ssigma;
  const double bbeta         = params.bbeta;
  const int T               = params.T;
  const double r             = params.r;
  const double w             = params.w;

  // Recover state variables from indices
  const int ix  = blockIdx.x * blockDim.x + threadIdx.x;
  const int ie  = threadIdx.y;

  double expected;
  double utility;
  double cons;
  double VV = pow(-10.0,5.0);

  for(int ixp = 0; ixp < nx; ixp++){

    expected = 0.0;
    if(age < T-1){
      for(int iep = 0; iep < ne; iep++){
        expected = expected + P[ie*ne + iep]*V[(age+1)*nx*ne + ixp*ne + iep];
      }
    }

    cons  = (1 + r)*xgrid[ix] + egrid[ie]*w - xgrid[ixp];

    utility = pow(cons, 1-ssigma) / (1-ssigma) + bbeta*expected;

    if(cons <= 0){
      utility = pow(-10.0, 5.0);
    }

    if(utility >= VV){
      VV = utility;
    }

    utility = 0.0;
  }

  V[age*nx*ne + ix*ne + ie] = VV;
}



int main()
{
  // Grids
  const int nx              = 300; 
  const double xmin          = 0.1;
  const double xmax          = 4.0;
  const int ne              = 15;
  const double ssigma_eps    = 0.02058;
  const double llambda_eps   = 0.99;
  const double m             = 1.5;

  // Parameters
  const double ssigma        = 2;
  const double eeta          = 0.36;
  const double ppsi          = 0.89;
  const double rrho          = 0.5;
  const double llambda       = 1;
  const double bbeta         = 0.97;
  const int T             	= 10;

  // Prices
  const double r             = 0.07;
  const double w             = 5;

  parameters params = {nx, xmin, xmax, ne, ssigma_eps, llambda_eps, m, ssigma, eeta, ppsi, rrho, llambda, bbeta, T, r, w};

  // Pointers to variables in the DEVICE memory
  double *V, *X, *E, *P;
  size_t sizeX = nx*sizeof(double);
  size_t sizeE = ne*sizeof(double);
  size_t sizeP = ne*ne*sizeof(double);
  size_t sizeV = T*ne*nx*sizeof(double);

  hipMalloc((void**)&X, sizeX);
  hipMalloc((void**)&E, sizeE);
  hipMalloc((void**)&P, sizeP);
  hipMalloc((void**)&V, sizeV);

  // Parameters for CUDA: cada block tiene ne columnas, y una fila que representa un valor de x
  //                      Hay nx blocks
  //                      Cada layer es una edad >= hay 80 layers

  const int block_size = 30;
  dim3 dimBlock(block_size, ne);
  dim3 dimGrid(nx/block_size, 1);


  // Variables in the host have "h" prefix
  // I create the grid for X
  double hxgrid[nx];
  gridx(nx, xmin, xmax, hxgrid);

  // I create the grid for E and the probability matrix
  double hegrid[ne];
  double hP[ne*ne];
  gride(ne, ssigma_eps, llambda_eps, m, hegrid);
  eprob(ne, ssigma_eps, llambda_eps, m, hegrid, hP);

    // Exponential of the grid e
  for(int i=0; i<ne; i++){
    hegrid[i] = exp(hegrid[i]);
  }

  double *hV;
  hV = (double *)malloc(sizeV);

  // Copy matrices from host (CPU) to device (GPU) memory
  hipMemcpy(X, hxgrid, sizeX, hipMemcpyHostToDevice);
  hipMemcpy(E, hegrid, sizeE, hipMemcpyHostToDevice);
  hipMemcpy(P, hP, sizeP, hipMemcpyHostToDevice);
  hipMemcpy(V, hV, sizeV, hipMemcpyHostToDevice);

  std::cout << " " << std::endl;
  std::cout << "Life cycle computation: " << std::endl;
  std::cout << " " << std::endl;

  // Time the GPU startup overhead
  clock_t t;
  clock_t t0;
  t0 	= clock();
  t 	= t0;

  for(int age=T-1; age>=0; age--){
    Vmaximization<<<dimGrid,dimBlock>>>(params, X, E, P, age, V);
    hipDeviceSynchronize();

  	t = clock() - t0;
  	std::cout << "Age: " << age << ". Time: " << ((double)t)/CLOCKS_PER_SEC << " seconds." << std::endl;

  }

  std::cout << " " << std::endl;
  t = clock() - t0;
  std::cout << "TOTAL ELAPSED TIME: " << ((double)t)/CLOCKS_PER_SEC << " seconds. " << std::endl;

  hipMemcpy(hV, V, sizeV, hipMemcpyDeviceToHost);

  // Free variables in device memory
  hipFree(V);
  hipFree(X);
  hipFree(E);
  hipFree(P);

  std::cout << " " << std::endl;
  std::cout << " - - - - - - - - - - - - - - - - - - - - - " << std::endl;
  std::cout << " " << std::endl;
  std::cout << "The first entries of the value function: " << std::endl;
  std::cout << " " << std::endl;

  for(int i = 0; i<3; i++){
    std::cout << hV[i] << std::endl;
  }

  std::cout << " " << std::endl;

  return 0;
}
